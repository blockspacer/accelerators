#include "hip/hip_runtime.h"
#include "float_ops.h"

#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <algorithm> 

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

using std::vector;
using std::equal;
using std::for_each;

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

extern void scan_hillis_single_block(
  float * d_out, const float * const d_in, const int size);

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(EXIT_FAILURE);
  }
}

float rand_logic_value() 
{
  return rand() % 2;
}

int main(int argc, char **argv)
{
  /// Check device
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "error: no devices supporting CUDA.\n");
      exit(EXIT_FAILURE);
  }
  int dev = 0;
  hipSetDevice(dev);

  hipDeviceProp_t devProps;
  if (hipGetDeviceProperties(&devProps, dev) == 0)
  {
      printf("Using device %d:\n", dev);
      printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
             devProps.name, (int)devProps.totalGlobalMem, 
             (int)devProps.major, (int)devProps.minor, 
             (int)devProps.clockRate);
  }
  
  int whichKernel = 0;
  if (argc == 2) {
      whichKernel = atoi(argv[1]);
  }

  /// Real work
  const int maxThreadsPerBlock = 8;
  const int kArraySize = maxThreadsPerBlock * 2 - 1;
  const int KBytesInArray = kArraySize * sizeof(float);

  // Serial:
  // generate the input array on the host
  float h_in[kArraySize];
  vector<float> h_gold;
  vector<float> h_out(kArraySize, 0);
  float sum = 0;
  for(int i = 0; i < kArraySize; i++) {
    h_gold.push_back(sum);
    float tmp = i+1;
    h_in[i] = tmp;
    sum += tmp;
  }
  
  // Parallel
  // declare GPU memory pointers
  float * d_in, * d_out, * d_predicat;
  {
    // allocate GPU memory
    hipMalloc((void **) &d_in, KBytesInArray);
    hipMalloc((void **) &d_out, KBytesInArray); // overallocated

    // transfer the input array to the GPU
    checkCudaErrors(hipMemcpy(d_in, h_in, KBytesInArray, hipMemcpyHostToDevice)); 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    switch(whichKernel) {
	    case 0:
			printf("Running reduce hill exclusive\n");
			hipEventRecord(start, 0);
			scan_hillis_single_block(d_in, d_out, kArraySize);
			checkCudaErrors(hipGetLastError());
			hipEventRecord(stop, 0);
			break;
	    default:
			fprintf(stderr, "error: ran no kernel\n");
			exit(EXIT_FAILURE);
    }
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);    
    elapsedTime /= 100.0f;      // 100 trials

    // copy back the sum from GPU
    checkCudaErrors(hipMemcpy(&h_out[0], d_out, KBytesInArray, hipMemcpyDeviceToHost));
    
    printf("average time elapsed: %f\n", elapsedTime);

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);
  }
  
  /// Check result
  assert(h_out.size() == h_gold.size());
  // раз значения uint можно просто проверить оператором ==
  assert(equal(h_gold.begin(), h_gold.end(), h_out.begin()
  //, AlmostEqualPredicate
  ));
  return 0;
}

