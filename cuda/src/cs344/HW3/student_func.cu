#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Definition Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

// C
#include <float.h>
#include <stdio.h>

// reuse
#include "utils.h"

const int maxThreadsPerBlock = 1024;


template <class Type> __device__ Type min_cuda( Type a, Type b ) {
  // I - +inf
  return a < b ? a : b;
}

template <class Type> __device__ Type max_cuda( Type a, Type b ) {
  // I - -inf
  return a > b ? a : b;
}

inline int isPow2(int a) {
  return !(a&(a-1));
}

class ReduceOperation {
public:
  virtual ~ReduceOperation() {}
  __device__ 
  virtual float operator()(float a, float b) const = 0;
  __device__
  virtual float I() const = 0;
};

class ComparatorMax : public ReduceOperation {
public:
  __device__ 
  virtual float operator()(float a, float b) const {
    return max_cuda<float>(a, b);
  }
  
  ComparatorMax() : I_val(-FLT_MAX) {}
  //explicit ComparatorMax(float value) : I_val(value) {}
  
  __device__
  virtual float I() const {
    return I_val;
  }
private:
  const float I_val;
};

__global__ void shmem_max_reduce_kernel(
    float * d_out, 
    const float * d_in /*для задания важна константность*/,
    const int size/*, const ReduceOperation* const op*/)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];
    
    //op->I();  // no way
    
    //float I = 
    

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // load shared mem from global mem
    if (myId < size)
      sdata[tid] = d_in[myId];
    else {
      // заполняем нейтральными элементами
      sdata[tid] = 
      //op->I();  // no way
      -FLT_MAX;
    }
    __syncthreads();            // make sure entire block is loaded!
    
    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
          float tmp =  
	    max_cuda<float>
	    //(*op)
	    (sdata[tid], sdata[tid + s]); 
	  sdata[tid] = tmp;
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}

void reduce_shared(float const * const d_in, float * const d_out, int size, const ReduceOperation* const op) 
{
  int threads = maxThreadsPerBlock;
  int blocks = ceil((1.0f*size) / maxThreadsPerBlock);
  int ARRAY_BYTES = size * sizeof(float);
  
  // assumes that size is not greater than maxThreadsPerBlock^2
  // and that size is a multiple of maxThreadsPerBlock
  assert(size <= threads * threads);  // для двушаговой редукции, чтобы уложиться
  assert(blocks * threads >= size);  // нужно будет ослабить - shared-mem дозаполним внутри ядер
  assert(isPow2(threads));  // должно делиться на 2 до конца
  
  float * d_intermediate;  // stage 1 result
  hipMalloc((void **) &d_intermediate, ARRAY_BYTES); // overallocated

  // Step 1: Вычисляем результаты для каждого блока
  // TODO: Error!!! "Segfault"
  shmem_max_reduce_kernel<<<blocks, threads, threads * sizeof(float)>>>(d_intermediate, d_in, size/*, op*/);
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());

  // Step 2: Комбинируем разультаты блоков и это ограничение на размер входных данных
  // now we're down to one block left, so reduce it
  threads = blocks; // launch one thread for each block in prev step
  blocks = 1;
  //shmem_max_reduce_kernel<<<blocks, threads, threads * sizeof(float)>>>(d_out, d_intermediate, threads, op);
  
  hipFree(d_intermediate);
}

// TODO: нужны временные буфферы

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  
 
  
  
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
       
       массив с данными должен быть не изменным, поэтому нужно хранить копию в shared
    */
  
  float* d_elem;
  float h_elem;
  hipMalloc((void **) &d_elem, sizeof(float));  // 1 значение
  
  ComparatorMax op;
  reduce_shared(d_logLuminance, d_elem, numRows * numCols, &op);
  
  hipFree(d_elem);
  
  /*
    2) subtract them to find the range
    
    // Похоже гистограмма как таковая не нужна
    // TODO: Можно ли использовать cdf? кажется можно
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */


}
