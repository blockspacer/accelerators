#include "hip/hip_runtime.h"
// TODO: сделать min and max reduce not in place

// C
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <float.h>

// C++
#include <vector>
#include <algorithm>

// 3rdparty
#include <hip/hip_runtime.h>
const int maxThreadsPerBlock = 1024;

// http://habrahabr.ru/post/146793/ !! трюки на С++

// Scan: 
// 1. Serial reguces - проблема в том, что если использовать reduce из лекции, то он портит исходный массив.
//   а значить нужны локальные копии для каждого потока. Work in place.
//   http://stackoverflow.com/questions/2187189/creating-arrays-in-nvidia-cuda-kernel - может потребоватся огромная память.
//
// 2.
//
// 3.
//
// http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html

// Float comparison http://floating-point-gui.de/errors/comparison/ - Java sample
// http://www.parashift.com/c++-faq/floating-point-arith.html
// http://docs.oracle.com/cd/E19957-01/806-3568/ncg_goldberg.html - матчасть
#include <cmath>  /* for std::abs(double) */

// не коммутативное
// isEqual(x,y) != isEqual(y,x)
inline bool isEqual(float x, float y)
{
  const float epsilon = 1e-2;/* some small number such as 1e-5 */;
  //printf("Delta = %f\n", x -y);
  //printf("x = %f\n", x);
  //printf("y = %f\n", y);
  return std::abs(x - y) <= epsilon * std::abs(x);
  // see Knuth section 4.2.2 pages 217-218
}

inline int isPow2(int a) {
  return !(a&(a-1));
}

// http://valera.asf.ru/cpp/book/c10.html
//#define max_cuda( a, b ) ( ((a) > (b)) ? (a) : (b) )
//#define min_cuda( a, b ) ( ((a) < (b)) ? (a) : (b) )

// Нейтральные элементы
// http://stackoverflow.com/questions/2684603/how-do-i-initialize-a-float-to-its-max-min-value

template <class Type> __device__ Type min_cuda( Type a, Type b ) {
  // I - +inf
  return a < b ? a : b;
}

template <class Type> __device__ Type max_cuda( Type a, Type b ) {
  // I - -inf
  return a > b ? a : b;
}

using std::vector;

__global__ void shmem_max_reduce_kernel(
    float * d_out, 
    const float * d_in /*для задания важна константность*/,
    const int size)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // load shared mem from global mem
    if (myId < size)
      sdata[tid] = d_in[myId];
    else {
      // заполняем нейтральными элементами
      sdata[tid] = -FLT_MAX;
    }
    
    __syncthreads();            // make sure entire block is loaded!
    
    //assert(isPow2(blockDim.x));  // нельзя

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
          float tmp =  max_cuda<float>(sdata[tid], sdata[tid + s]); 
	  sdata[tid] = tmp;
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}

__global__ void shmem_min_reduce_kernel(
    float * d_out, 
    const float * d_in /*для задания важна константность*/, int size)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // load shared mem from global mem
    if (myId < size)
      sdata[tid] = d_in[myId];
    else {
      // заполняем нейтральными элементами
      sdata[tid] = +FLT_MAX;
    }
    
    __syncthreads();            // make sure entire block is loaded!

    // do reduction in shared mem
    //TODO: blockDim должна быть степенью 2
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
          float tmp =  min_cuda<float>(sdata[tid], sdata[tid + s]); 
	  sdata[tid] = tmp;
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}

//TODO: не хотелось писать в сигнатуру, хотя удобство сомнительно
template<bool isMin>  
void reduce_shared_min(
    float * const d_out, 
    float * const d_intermediate, float const * const d_in, 
    int size
    //, bool isMin
    ) 
{
  int threads = maxThreadsPerBlock;
  int blocks = size / threads;  // отбрасываем дробную часть
  
  // assumes that size is not greater than maxThreadsPerBlock^2
  // and that size is a multiple of maxThreadsPerBlock
  assert(size <= threads * threads);  // для двушаговой редукции, чтобы уложиться
  //assert(blocks * threads == size);  // нужно будет ослабить - shared-mem дозаполним внутри ядер
  assert(isPow2(threads));  // должно делиться на 2 до конца

  // Step 1: Вычисляем результаты для каждого блока
  if (isMin)
    shmem_min_reduce_kernel<<<blocks, threads, threads * sizeof(float)>>>(d_intermediate, d_in, size);
  else {
    shmem_max_reduce_kernel<<<blocks, threads, threads * sizeof(float)>>>(d_intermediate, d_in, size);
  }

  // Step 2: Комбинируем разультаты блоков и это ограничение на размер входных данных
  // now we're down to one block left, so reduce it
  threads = blocks; // launch one thread for each block in prev step
  blocks = 1;
  if (isMin)
    shmem_min_reduce_kernel<<<blocks, threads, threads * sizeof(float)>>>(d_out, d_intermediate, threads);
  else {
    shmem_max_reduce_kernel<<<blocks, threads, threads * sizeof(float)>>>(d_out, d_intermediate, threads);
  }
}

int main(int argc, char **argv)
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "error: no devices supporting CUDA.\n");
      exit(EXIT_FAILURE);
  }
  int dev = 0;
  hipSetDevice(dev);

  hipDeviceProp_t devProps;
  if (hipGetDeviceProperties(&devProps, dev) == 0)
  {
      printf("Using device %d:\n", dev);
      printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
	      devProps.name, (int)devProps.totalGlobalMem, 
	      (int)devProps.major, (int)devProps.minor, 
	      (int)devProps.clockRate);
  }

  const int ARRAY_SIZE = (1 << 19);  //TODO: важно правильно выбрать
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // generate the input array on the host
  float h_in[ARRAY_SIZE];
  for(int i = 0; i < ARRAY_SIZE; i++) {
      // generate random float in [-1.0f, 1.0f]
      h_in[i] = -1.0f + (float)random()/((float)RAND_MAX/2.0f);
  }
  h_in[ARRAY_SIZE-1] = -1000.0;
  h_in[0] = 1000.0;
  
  // Ищем минимум
  // http://stackoverflow.com/questions/259297/how-do-you-copy-the-contents-of-an-array-to-a-stdvector-in-c-without-looping
  vector<float> hIn;
  unsigned dataArraySize = sizeof(h_in) / sizeof(float);
  assert(dataArraySize == ARRAY_SIZE);
  hIn.insert(hIn.end(), &h_in[0], &h_in[dataArraySize]);
  assert(hIn.size() == ARRAY_SIZE);
  
  // Используем стандартную функцию
  // http://stackoverflow.com/questions/8340569/stdvector-and-stdmin-behavior 
  // Похоже можно искать сразу в векторе
  float serialMin = *std::min_element(hIn.begin(),hIn.end());
  float serialMax = *std::max_element(hIn.begin(),hIn.end());


  // declare GPU memory pointers
  float * d_in;
  float * d_intermediate;  // stage 1 result
  float * d_out;

  // allocate GPU memory
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_intermediate, ARRAY_BYTES); // overallocated
  hipMalloc((void **) &d_out, sizeof(float));  // 1 значение

  // transfer the input array to the GPU
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice); 

  int whichKernel = 0;
  if (argc == 2) {
      whichKernel = atoi(argv[1]);
  }
    
  {     
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // launch the kernel
    switch(whichKernel) {
    case 0:
	printf("Running min reduce with shared mem\n");
	hipEventRecord(start, 0);
	for (int i = 0; i < 100; i++)
	{
	    reduce_shared_min<true>(d_out, d_intermediate, d_in, ARRAY_SIZE);//, true);
	}
	hipEventRecord(stop, 0);
	break;
    default:
	fprintf(stderr, "error: ran no kernel\n");
	exit(EXIT_FAILURE);
    }
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);    
    elapsedTime /= 100.0f;      // 100 trials

    // copy back the sum from GPU
    float h_out;
    hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);
    
    assert(isEqual(h_out, serialMin));
    printf("average time elapsed: %f\n", elapsedTime);
  }
  
  // MAX
  {     
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // launch the kernel
    switch(whichKernel) {
    case 0:
	printf("Running min reduce with shared mem\n");
	hipEventRecord(start, 0);
	//for (int i = 0; i < 100; i++)
	//{
	    reduce_shared_min<false>(d_out, d_intermediate, d_in, ARRAY_SIZE);//, false);
	//}
	hipEventRecord(stop, 0);
	break;
    default:
	fprintf(stderr, "error: ran no kernel\n");
	exit(EXIT_FAILURE);
    }
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);    
    elapsedTime /= 100.0f;      // 100 trials

    // copy back the sum from GPU
    float h_out;
    hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);
    
    assert(isEqual(h_out, serialMax));
    printf("average time elapsed: %f\n", elapsedTime);
  }

  // free GPU memory allocation
  hipFree(d_in);
  hipFree(d_intermediate);
  hipFree(d_out);
  return 0;
}
