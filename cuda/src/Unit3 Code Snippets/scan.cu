#include "hip/hip_runtime.h"
// C
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// C++
#include <iostream>

// 3rdparty
#include <hip/hip_runtime.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    //assert(false && "CUDA error");
    exit(1);
  }
}


// Scan: 
// 1. Serial reguces - проблема в том, что если использовать reduce из лекции, то он портит исходный массив.
//   а значить нужны локальные копии для каждого потока. Work in place.
//   http://stackoverflow.com/questions/2187189/creating-arrays-in-nvidia-cuda-kernel - может потребоватся огромная память.
//
// 2.
//
// 3.
//
// http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html

const int maxThreadsPerBlock = 1024;

/*
// serial:
// TODO: причем тут f(elem)?
{
  out[0] = 0;
  for j from 1 to n do
    out[j] = out[j-1] + f(in[j-1]);
}
*/

/*
// Hillis and Steele
// parallel with one buffer:
// TODO: не понял в чем проблема, но похоже она в синхронизации
//   хотя нет, похоже дело в том что расчет in-place. Нет не в этом дело.
//   На стадиях обработки данные затираются.
for d = 1 to log2(n) do
  for all k in parallel do
    if k >= 2^d then
      x[k] = x[k - 2^(d-1)] + x[k]

// parallel separated in and out buffers:
for d = 1 to log2(n) do
  for all k in parallel do
    if k >= 2^d then
      x[out][k] = x[in][k-2^(d-1)] + x[in][k]
    else
      x[out][k] = x[in][k]
*/

// http://www.cplusplus.com/reference/algorithm/swap/
__device__ void cuSwap(int& a, int& b) 
{
  int c(a); a=b; b=c;
}

__global__ void global_scan_kernel_one_block(float * d_out, const float * const d_in, int n)
{
  //int myId = threadIdx.x + blockDim.x * blockIdx.x;  // not one block!
  
  // результаты работы потоков можем расшаривать через эту
  // память или через глобальную
  extern __shared__ float temp[];  
  int localId  = threadIdx.x;
  
  if (localId >= n) 
    return;
  
  //  for
  int p_sink = 0;  // int1 не работает
  int p_source = 1;

  // Load input into shared memory.  
  // This is exclusive scan, so shift right by one  
  // and set first element to 0  
  temp[p_sink * n + localId] = (localId > 0) ? d_in[localId-1] : 0;  
  __syncthreads();  

  /*for (int offset = 1; offset < n; offset *= 2)  // 2^i
  {  
    cuSwap(p_sink, p_source);
    
    if (localId >= offset)  
      temp[p_sink * n+localId] += temp[p_source * n + localId - offset];  
    else  
      temp[p_sink * n+localId] = temp[p_source * n+localId];  
    
    // буффера переписали
    __syncthreads();  
  }  */

  // p_sink == 0?
  // Пишем из текущего буффера
  d_out[localId] = temp[p_sink * n + localId /*1*/]; // write output 
}

void scan_hillis(/*float * d_out,*/ float * d_intermediate, float * d_in, int size) 
{
  // Precond:
  // assumes that size is not greater than maxThreadsPerBlock^2
  // and that size is a multiple of maxThreadsPerBlock

  int threads = maxThreadsPerBlock;
  int blocks = 1;//size / maxThreadsPerBlock;
  assert(blocks == 1);  // TODO: пока чтобы не комбинировать результаты блоков

  global_scan_kernel_one_block<<<blocks, threads, threads * sizeof(float) * 2>>>(d_intermediate, d_in, size);
}

int main(int argc, char **argv)
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "error: no devices supporting CUDA.\n");
      exit(EXIT_FAILURE);
  }
  int dev = 0;
  hipSetDevice(dev);

  hipDeviceProp_t devProps;
  if (hipGetDeviceProperties(&devProps, dev) == 0)
  {
      printf("Using device %d:\n", dev);
      printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
             devProps.name, (int)devProps.totalGlobalMem, 
             (int)devProps.major, (int)devProps.minor, 
             (int)devProps.clockRate);
  }

  const int ARRAY_SIZE = 8;//maxThreadsPerBlock;//1 << 20;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // Serial:
  // generate the input array on the host
  float h_in[ARRAY_SIZE];
  float sum = 0.0f;
  for(int i = 0; i < ARRAY_SIZE; i++) {
    printf("%f, ", sum);  
    // generate random float in [-1.0f, 1.0f]
    h_in[i] = 1.0f * (i+1);// + (float)random()/((float)RAND_MAX/2.0f);
    sum += h_in[i];  
  }
  printf("\n");//, sum);
  

  // Parallel
  // declare GPU memory pointers
  float * d_in, * d_intermediate;//, * d_out;

  // allocate GPU memory
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_intermediate, ARRAY_BYTES); // overallocated
  //hipMalloc((void **) &d_out, sizeof(float));

  // transfer the input array to the GPU
  checkCudaErrors(hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice)); 

  int whichKernel = 0;
  if (argc == 2) {
      whichKernel = atoi(argv[1]);
  }
      
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  whichKernel = 0;
  switch(whichKernel) {
  case 0:
      printf("Running global reduce\n");
      hipEventRecord(start, 0);
      scan_hillis(/*d_out,*/ d_intermediate, d_in, ARRAY_SIZE);//, false);
      checkCudaErrors(hipGetLastError());
      hipEventRecord(stop, 0);
      break;
  default:
      fprintf(stderr, "error: ran no kernel\n");
      exit(EXIT_FAILURE);
  }
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);    
  elapsedTime /= 100.0f;      // 100 trials

  // copy back the sum from GPU
  float h_out[ARRAY_SIZE]; // ARRAY_BYTES
  checkCudaErrors(hipMemcpy(h_out, d_intermediate, ARRAY_BYTES, hipMemcpyDeviceToHost));
  for (int i = 0; i < ARRAY_SIZE; ++i) {
    printf("%f, ", h_out[i]);
  }
  printf("\n");//, sum);

  //printf("Parallel sum: %f\n", h_out);

  printf("average time elapsed: %f\n", elapsedTime);

  // free GPU memory allocation
  hipFree(d_in);
  hipFree(d_intermediate);
  //hipFree(d_out);
      
  return 0;
}
