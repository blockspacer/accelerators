#include "hip/hip_runtime.h"
 #include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Scan: 
// 1. Serial reguces - проблема в том, что если использовать reduce из лекции, то он портит исходный массив.
//   а значить нужны локальные копии для каждого потока. Work in place.
//   http://stackoverflow.com/questions/2187189/creating-arrays-in-nvidia-cuda-kernel - может потребоватся огромная память.
//
// 2.
//
// 3.
//
// http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html

const int maxThreadsPerBlock = 1024;

/*
// serial:
// TODO: причем тут f(elem)?
{
  out[0] = 0;
  for j from 1 to n do
    out[j] = out[j-1] + f(in[j-1]);
}
*/

/*
// Hillis and Steele
// parallel with one buffer:
// TODO: не понял в чем проблема, но похоже она в синхронизации
//   хотя нет, похоже дело в том что расчет in-place
for d = 1 to log2(n) do
  for all k in parallel do
    if k >= 2^d then
      x[k] = x[k - 2^(d-1)] + x[k]

// parallel separated in and out buffers:
for d = 1 to log2(n) do
  for all k in parallel do
    if k >= 2^d then
      x[out][k] = x[in][k-2^(d-1)] + x[in][k]
    else
      x[out][k] = x[in][k]
*/

__global__ void global_scan_kernel_one_block(float * d_out, float * d_in)
{
//    int myId = threadIdx.x + blockDim.x * blockIdx.x;  // not one block!
    int tid  = threadIdx.x;
}

void scan(float * d_out, float * d_intermediate, float * d_in, int size) 
{
  // Precond:
  // assumes that size is not greater than maxThreadsPerBlock^2
  // and that size is a multiple of maxThreadsPerBlock

  int threads = maxThreadsPerBlock;
  int blocks = 1;//size / maxThreadsPerBlock;
  printf("Count blocks: %d\n", blocks);
  global_scan_kernel_one_block<<<blocks, threads>>>(d_intermediate, d_in);
}

int main(int argc, char **argv)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    hipSetDevice(dev);

    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
               devProps.name, (int)devProps.totalGlobalMem, 
               (int)devProps.major, (int)devProps.minor, 
               (int)devProps.clockRate);
    }

    const int ARRAY_SIZE = maxThreadsPerBlock;//1 << 20;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    float sum = 0.0f;
    for(int i = 0; i < ARRAY_SIZE; i++) {
        // generate random float in [-1.0f, 1.0f]
        h_in[i] = -1.0f + (float)random()/((float)RAND_MAX/2.0f);
        sum += h_in[i];
    }

    // declare GPU memory pointers
    float * d_in, * d_intermediate, * d_out;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_intermediate, ARRAY_BYTES); // overallocated
    hipMalloc((void **) &d_out, sizeof(float));

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice); 

    int whichKernel = 0;
    if (argc == 2) {
        whichKernel = atoi(argv[1]);
    }
        
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // launch the kernel
  int countTries = 1;
    switch(whichKernel) {
    case 0:
        printf("Running global reduce\n");
        hipEventRecord(start, 0);
        for (int i = 0; i < countTries; i++)
        {
            scan(d_out, d_intermediate, d_in, ARRAY_SIZE);//, false);
        }
        hipEventRecord(stop, 0);
        break;
    case 1:
        printf("Running reduce with shared mem\n");
        hipEventRecord(start, 0);
        for (int i = 0; i < countTries; i++)
        {
            scan(d_out, d_intermediate, d_in, ARRAY_SIZE);//, true);
        }
        hipEventRecord(stop, 0);
        break;
    default:
        fprintf(stderr, "error: ran no kernel\n");
        exit(EXIT_FAILURE);
    }
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);    
    elapsedTime /= 100.0f;      // 100 trials

    // copy back the sum from GPU
    float h_out;
    hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);

    printf("average time elapsed: %f\n", elapsedTime);

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_intermediate);
    hipFree(d_out);
        
    return 0;
}
