#include "hip/hip_runtime.h"
 // TODO: расширить на несколько блоков
// Scan: 
// http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html


// C
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// C++
#include <iostream>
#include <vector>
#include <algorithm> 

// 3rdparty
#include <hip/hip_runtime.h>

// App
#include "float_ops.h"

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    //assert(false && "CUDA error");
    exit(1);
  }
}


const int maxThreadsPerBlock = 1024;

using std::vector;
using std::equal;
using std::for_each;

int main(int argc, char **argv)
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "error: no devices supporting CUDA.\n");
      exit(EXIT_FAILURE);
  }
  int dev = 0;
  hipSetDevice(dev);

  hipDeviceProp_t devProps;
  if (hipGetDeviceProperties(&devProps, dev) == 0)
  {
      printf("Using device %d:\n", dev);
      printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
             devProps.name, (int)devProps.totalGlobalMem, 
             (int)devProps.major, (int)devProps.minor, 
             (int)devProps.clockRate);
  }

  const int ARRAY_SIZE = maxThreadsPerBlock * 7 - 4;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // Serial:
  // generate the input array on the host
  float h_in[ARRAY_SIZE];
  float h_scan_gold[ARRAY_SIZE];
  float sum = 0.0f;
  for(int i = 0; i < ARRAY_SIZE; i++) {
    h_scan_gold[i] = sum;
    h_in[i] = 1.0f * (i+1);
    sum += h_in[i];  
  }

  // Parallel
  // declare GPU memory pointers
  float * d_in, * d_out;//, * d_out;

  // allocate GPU memory
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES); // overallocated
  //hipMalloc((void **) &d_out, sizeof(float));

  // transfer the input array to the GPU
  checkCudaErrors(hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice)); 

  int whichKernel = 0;
  if (argc == 2) {
      whichKernel = atoi(argv[1]);
  }
      
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  whichKernel = 0;
  switch(whichKernel) {
  case 0:
      printf("Running reduce hill exclusive\n");
      hipEventRecord(start, 0);
      //scan_hillis_single_block(d_out, d_in, ARRAY_SIZE);
      checkCudaErrors(hipGetLastError());
      hipEventRecord(stop, 0);
      break;
  default:
      fprintf(stderr, "error: ran no kernel\n");
      exit(EXIT_FAILURE);
  }
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);    
  elapsedTime /= 100.0f;      // 100 trials

  // copy back the sum from GPU
  float h_out[ARRAY_SIZE]; // ARRAY_BYTES
  checkCudaErrors(hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
  
  printf("average time elapsed: %f\n", elapsedTime);

  // free GPU memory allocation
  hipFree(d_in);
  hipFree(d_out);
  
  // Check: сравнить бы с моделью
  vector<float> hGold;
  vector<float> hOut;
  unsigned dataArraySize = sizeof(h_scan_gold) / sizeof(float);
  assert(dataArraySize == ARRAY_SIZE);
  hGold.insert(hGold.end(), &h_scan_gold[0], &h_scan_gold[dataArraySize]);
  hOut.insert(hOut.end(), &h_out[0], &h_out[dataArraySize]);
  assert(hOut.size() == hGold.size());
  assert(
  equal
  //for_each
    //equal_adapt
    (hGold.begin(), hGold.end(), hOut.begin(), AlmostEqualPredicate)//;
  );
  
     
  return 0;
}

