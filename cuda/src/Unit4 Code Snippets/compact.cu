#include "hip/hip_runtime.h"
 // TODO: расширить на несколько блоков
// Scan: 
// http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html


// C
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// C++
#include <iostream>
#include <vector>
#include <algorithm> 

// 3rdparty
#include <hip/hip_runtime.h>

// App
#include "float_ops.h"

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

extern void scan_hillis_single_block(const unsigned int * const d_in, unsigned int * const d_out, const int size);

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    //assert(false && "CUDA error");
    exit(1);
  }
}

using std::vector;
using std::equal;
using std::for_each;

int main(int argc, char **argv)
{
  /// Check device
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "error: no devices supporting CUDA.\n");
      exit(EXIT_FAILURE);
  }
  int dev = 0;
  hipSetDevice(dev);

  hipDeviceProp_t devProps;
  if (hipGetDeviceProperties(&devProps, dev) == 0)
  {
      printf("Using device %d:\n", dev);
      printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
             devProps.name, (int)devProps.totalGlobalMem, 
             (int)devProps.major, (int)devProps.minor, 
             (int)devProps.clockRate);
  }
  
  int whichKernel = 0;
  if (argc == 2) {
      whichKernel = atoi(argv[1]);
  }

  /// Real work
  const int maxThreadsPerBlock = 8;
  const int ARRAY_SIZE = maxThreadsPerBlock * 2 - 1;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned int);

  // Serial:
  // generate the input array on the host
  unsigned int h_in[ARRAY_SIZE];
  unsigned int h_scan_gold[ARRAY_SIZE];
  unsigned int sum = 0;
  for(int i = 0; i < ARRAY_SIZE; i++) {
    h_scan_gold[i] = sum;
    h_in[i] = i+1;
    sum += h_in[i];  
  }

  // Parallel
  // declare GPU memory pointers
  unsigned int * d_in, * d_out;//, * d_out;

  // allocate GPU memory
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES); // overallocated

  // transfer the input array to the GPU
  checkCudaErrors(hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice)); 
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  switch(whichKernel) {
  case 0:
      printf("Running reduce hill exclusive\n");
      hipEventRecord(start, 0);
      scan_hillis_single_block(d_in, d_out, ARRAY_SIZE);
      checkCudaErrors(hipGetLastError());
      hipEventRecord(stop, 0);
      break;
  default:
      fprintf(stderr, "error: ran no kernel\n");
      exit(EXIT_FAILURE);
  }
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);    
  elapsedTime /= 100.0f;      // 100 trials

  // copy back the sum from GPU
  unsigned int h_out[ARRAY_SIZE]; // ARRAY_BYTES
  checkCudaErrors(hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
  
  printf("average time elapsed: %f\n", elapsedTime);

  // free GPU memory allocation
  hipFree(d_in);
  hipFree(d_out);
  
  /// Check result
  vector<unsigned int> hGold;
  vector<unsigned int> hOut;
  unsigned dataArraySize = sizeof(h_scan_gold) / sizeof(unsigned int);
  assert(dataArraySize == ARRAY_SIZE);
  hGold.insert(hGold.end(), &h_scan_gold[0], &h_scan_gold[dataArraySize]);
  hOut.insert(hOut.end(), &h_out[0], &h_out[dataArraySize]);
  assert(hOut.size() == hGold.size());
  assert(equal(hGold.begin(), hGold.end(), hOut.begin()
  //, AlmostEqualPredicate
  ));
  return 0;
}

